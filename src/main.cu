#include <QtWidgets>
#include <nvml.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s)!\n", __FILE__, __LINE__, result, hipGetErrorString(result)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

class GPUInfoWidget : public QWidget {
    Q_OBJECT

public:
    GPUInfoWidget(QWidget *parent = nullptr) : QWidget(parent) {
        QVBoxLayout *layout = new QVBoxLayout(this);
        
        // Başlık
        QLabel *titleLabel = new QLabel("<h2 style=\"color:white;\">Alp - GPU Bilgi Uygulaması</h2>");
        layout->addWidget(titleLabel);

        // Genel GPU Bilgileri
        gpuInfoLabel = new QLabel();
        updateGPUInfo(gpuInfoLabel);
        layout->addWidget(gpuInfoLabel);

        // CUDA Kernels Bilgileri
        QPushButton *kernelInfoButton = new QPushButton("CUDA Kernels Bilgisi");
        layout->addWidget(kernelInfoButton);
        connect(kernelInfoButton, &QPushButton::clicked, this, &GPUInfoWidget::showKernelInfo);

        // Özellikler
        setStyleSheet("background-color: #2b2b2b; color: white;");
    }

    void updateGPUInfo(QLabel *label) {
        nvmlInit();
        nvmlDevice_t device;
        nvmlDeviceGetHandleByIndex(0, &device); // Sadece bir GPU varsa
        char name[NVML_DEVICE_NAME_BUFFER_SIZE];
        nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE);
        nvmlMemory_t mem;
        nvmlDeviceGetMemoryInfo(device, &mem);
        nvmlUtilization_t util;
        nvmlDeviceGetUtilizationRates(device, &util);
        nvmlShutdown();

        QString info = QString("<b>GPU Adı:</b> %1<br>").arg(name);
        info += QString("<b>Bellek Kullanımı:</b> %1 MB / %2 MB<br>").arg(mem.used / 1024 / 1024).arg(mem.total / 1024 / 1024);
        info += QString("<b>% GPU Kullanımı:</b> %1<br>").arg(util.gpu);
        info += QString("<b>% Bellek Kullanımı:</b> %1<br>").arg(util.memory);

        label->setText(info);
    }

signals:
    void showKernelInfo();

private slots:
    void showKernelInfo() {
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        int *d_array;
        CUDA_CHECK(hipMalloc(&d_array, 1000 * sizeof(int)));

        CUDA_CHECK(hipEventRecord(start));
        kernel<<<1, 1>>>(d_array);
        CUDA_CHECK(hipEventRecord(stop));

        CUDA_CHECK(hipEventSynchronize(stop));

        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

        QLabel *kernelInfoLabel = new QLabel(QString("<h2 style=\"color:white;\">CUDA Kernel Zamanlaması</h2>"));
        kernelInfoLabel->append(QString("Kernel Çalışma Süresi: %1 ms").arg(milliseconds));
        QVBoxLayout *layout = qobject_cast<QVBoxLayout*>(this->layout());
        layout->addWidget(kernelInfoLabel);
        
        CUDA_CHECK(hipFree(d_array));
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

private:
    QLabel *gpuInfoLabel;
};

int main(int argc, char **argv) {
    QApplication app(argc, argv);

    GPUInfoWidget widget;
    widget.show();

    return app.exec();
}

#include "main.moc"
